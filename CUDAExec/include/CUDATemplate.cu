#include "hip/hip_runtime.h"
#include <assert.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "gtest/gtest.h"

#include "CUDALib.h"

__global__ void exec_kernel()
{
    printf("\n I am thread %d from exec_kernel\n", threadIdx.x);
}

TEST(Test, exe)
{
    exec_kernel<<<1, 1>>>();
    auto err = hipDeviceSynchronize();
    EXPECT_EQ(err, hipSuccess);    
}

TEST(Test, lib)
{
    CUDALib lib;
    lib.run();
    auto err = hipDeviceSynchronize();
    EXPECT_EQ(err, hipSuccess);
}

int main(int argc, char** argv)
{
    ::testing::InitGoogleTest(&argc, argv);

    return RUN_ALL_TESTS();
}
