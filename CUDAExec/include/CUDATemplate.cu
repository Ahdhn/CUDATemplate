#include "hip/hip_runtime.h"
#include <assert.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "gtest/gtest.h"

__global__ void testKernel()
{
    printf("\n I am thread %d\n", threadIdx.x);
}

TEST(Test, simple)
{
    testKernel<<<1, 1>>>();
    auto err = hipDeviceSynchronize();
    EXPECT_EQ(err, hipSuccess);
}

int main(int argc, char** argv)
{
    ::testing::InitGoogleTest(&argc, argv);

    return RUN_ALL_TESTS();
}
