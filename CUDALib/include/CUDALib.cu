#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>

#include "CUDALib.h"



namespace detail {
__global__ void lib_kernel()
{
    printf("\n I am thread %d from lib_kernel\n", threadIdx.x);
}
}  // namespace detail

void CUDALib::run()
{
    detail::lib_kernel<<<1, 1>>>();   
}