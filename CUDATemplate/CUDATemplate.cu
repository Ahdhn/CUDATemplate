#include <stdio.h>
#include <assert.h>
#include <hip/hip_runtime.h>


__global__ void testKernel()
{
    printf("\n I am thread %d", threadIdx.x);
}

int main(int argc, char **argv)
{     
    testKernel<<<1, 1>>>();
    hipDeviceSynchronize();
    return EXIT_SUCCESS;
}

